
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

class Tensor {
public:
  int shape_[4];
  float* hptr_;
  float* dptr_;

  Tensor() {
    shape_[0] = 0;
    shape_[1] = 0;
    shape_[2] = 0;
    shape_[3] = 0;
    hptr_ = NULL;
    dptr_ = NULL;
  }

  ~Tensor() {
    shape_[0] = 0;
    shape_[1] = 0;
    shape_[2] = 0;
    shape_[3] = 0;
    if(hptr_ != NULL) {
      free(hptr_);
      hptr_ = NULL;
    }
    if(dptr_ != NULL) {
      hipFree(dptr_);
      dptr_ = NULL;
    }
  } 
  
  bool read(char* fname) {
    FILE* f = fopen(fname, "r");
    if(f == NULL) {
      std::cerr << "ERROR: Could not open file: " << fname << "\n";
      return false;
    }
    fscanf(f, "%d\n%d\n%d\n%d\n", &shape_[0], &shape_[1], &shape_[2], &shape_[3]); 
    if(!(shape_[0] > 0 && shape_[1] > 0 && shape_[2] > 0 && shape_[3] > 0)) {
      std::cerr << "ERROR: Invalid Tensor Dimensions: " << shape_[0] << " " << shape_[1] << " " << shape_[2] << " " << shape_[3] << "\n";
      return false;
    }
    hptr_ = (float*)malloc(shape_[0]*shape_[1]*shape_[2]*shape_[3]*sizeof(float));
    if(hptr_ == NULL) {
      std::cerr << "ERROR: Could not allocate space on host\n";
      return false;
    }
    hipMalloc(&dptr_, shape_[0]*shape_[1]*shape_[2]*shape_[3]*sizeof(float));
    if(dptr_ == NULL) {
      std::cerr << "ERROR: Could not allocate space on device\n";
      return false;
    }
    for(int i = 0; i < shape_[0]*shape_[1]*shape_[2]*shape_[3]; i++) {
      fscanf(f, "%f\n", &hptr_[i]);
    }
    fclose(f);
    return true;
  }

  bool write(char* fname) {
    FILE* f = fopen(fname, "w");
    if(f == NULL) {
      std::cerr << "ERROR: Could not open file: " << fname << "\n";
      return false;
    }
    fprintf(f, "%d\n%d\n%d\n%d\n", shape_[0], shape_[1], shape_[2], shape_[3]); 
    for(int i = 0; i < shape_[0]*shape_[1]*shape_[2]*shape_[3]; i++) {
      fprintf(f, "%f\n", hptr_[i]);
    }
    fclose(f);
    return true;
  }
  
  bool create(int a, int b, int c, int d) {
    shape_[0] = a;
    shape_[1] = b;
    shape_[2] = c;
    shape_[3] = d;
    if(!(shape_[0] > 0 && shape_[1] > 0 && shape_[2] > 0 && shape_[3] > 0)) {
      std::cerr << "ERROR: Invalid Tensor Dimensions: " << shape_[0] << " " << shape_[1] << " " << shape_[2] << " " << shape_[3] << "\n";
      return false;
    }
    hptr_ = (float*)malloc(shape_[0]*shape_[1]*shape_[2]*shape_[3]*sizeof(float));
    if(hptr_ == NULL) {
      std::cerr << "ERROR: Could not allocate space on host\n";
      return false;
    }
    memset(hptr_, 0, shape_[0]*shape_[1]*shape_[2]*shape_[3]*sizeof(float));
    hipMalloc(&dptr_, shape_[0]*shape_[1]*shape_[2]*shape_[3]*sizeof(float));
    if(dptr_ == NULL) {
      std::cerr << "ERROR: Could not allocate space on device\n";
      return false;
    }
    hipMemset(dptr_, 0, shape_[0]*shape_[1]*shape_[2]*shape_[3]*sizeof(float));
    return true;
  }

  int size() {
    return shape_[0]*shape_[1]*shape_[2]*shape_[3];
  }

  void copyToDevice() {
    hipMemcpy(dptr_, hptr_, shape_[0]*shape_[1]*shape_[2]*shape_[3]*sizeof(float), hipMemcpyHostToDevice);
  }
  
  void copyToHost() {
    hipMemcpy(hptr_, dptr_, shape_[0]*shape_[1]*shape_[2]*shape_[3]*sizeof(float), hipMemcpyDeviceToHost);
  }
};

#define BLOCK 16

__global__ void forward_kernel(float *y, const float *x, const float *k, const int B, const int M, const int C, const int H, const int W, const int K) {
  const int radius = (K-1)/2;
  const int x_out = blockIdx.x*BLOCK + (threadIdx.x - radius);
  const int y_out = blockIdx.y*BLOCK + (threadIdx.y - radius);

  const int H_out = H - K + 1;
  const int W_out = W - K + 1;

#define y4d(i3, i2, i1, i0) y[(i3) * (M * H_out * W_out) + (i2) * (H_out * W_out) + (i1) * (W_out) + i0]
#define x4d(i3, i2, i1, i0) x[(i3) * (C * H * W) + (i2) * (H * W) + (i1) * (W) + i0]
#define k4d(i3, i2, i1, i0) k[(i3) * (C * K * K) + (i2) * (K * K) + (i1) * (K) + i0]
  if((x_out >= 0 && x_out < W_out) && (y_out >= 0 && y_out < H_out)) {
    if(((threadIdx.x - radius) >= 0 && (threadIdx.x - radius) < BLOCK) &&
       ((threadIdx.y - radius) >= 0 && (threadIdx.y - radius) < BLOCK)) {
      for(int b = 0; b < B; ++b) {
        for(int m = 0; m < M; m++) {
          y4d(b,m,y_out,x_out) = 0;
          for(int c = 0; c < C; c++) {
            for(int p = 0; p < K; p++) {
              for(int q = 0; q < K; q++) {
                y4d(b,m,y_out,x_out) += x4d(b,c,y_out+p,x_out+q) * k4d(m,c,p,q);
              }
            }
          }
        }
      }
    }
  }
#undef y4d
#undef x4d
#undef k4d
}

void forward(Tensor &y, const Tensor &x, const Tensor &w) {
  // Extract the tensor dimensions into B,M,C,H,W,K
  const int B = x.shape_[0];
  const int M = y.shape_[1];
  const int C = x.shape_[1];
  const int K = w.shape_[3];
  const int H = x.shape_[2];
  const int W = x.shape_[3];

  const int radius = (K-1)/2;
  const int block = BLOCK + 2*radius;
  
  // Set the kernel dimensions
  dim3 gridDim(ceil((float)(W-2*radius)/((float)BLOCK)), ceil((float)(H-2*radius)/((float)BLOCK)), 1);
  dim3 blockDim(block, block, 1);

  // Call the kernel
  forward_kernel<<<gridDim, blockDim>>>(y.dptr_,x.dptr_,w.dptr_, B,M,C,H,W,K);

  // Use MSHADOW_CUDA_CALL to check for CUDA runtime errors.
  hipDeviceSynchronize();
}

void generate_random() {
  Tensor w;
  Tensor x;
  Tensor w2;
  Tensor x2;

  w.create(12, 1, 5, 5);
  w2.create(24, 12, 5, 5);
  x.create(10000, 1, 70, 70);
  x2.create(10000, 12, 33, 33);

  for(int i = 0; i < x.size(); i++) {
    x.hptr_[i] = (float)(rand()%10000)/10000.0;
  }
  for(int i = 0; i < w.size(); i++) {
    w.hptr_[i] = (float)(rand()%10000)/10000.0;
  }
  for(int i = 0; i < x2.size(); i++) {
    x2.hptr_[i] = (float)(rand()%10000)/10000.0;
  }
  for(int i = 0; i < w2.size(); i++) {
    w2.hptr_[i] = (float)(rand()%10000)/10000.0;
  }

  w.write("w1.raw");
  w2.write("w2.raw");
  x.write("x.raw");
  x2.write("x2.raw");
}

int main(int argc, char **argv) {
  std::cout << argc << "\n";

  Tensor w;
  Tensor x;
  Tensor y;

  if(!y.create(10000, 12, 66, 66)) return -1;

  if(!x.read("data/x1.raw")) return -1;
  if(!w.read("data/w1.raw")) return -1;

  forward(y, x, w);
  y.copyToHost();

  if(!y.write("data/y1.raw")) return -1;
  return 0;
}
